// #include <stdio.h>
// #include <stdlib.h>
// #include <cuda_runtime.h>

// template<class T>
// struct SharedMemory {
//     __device__ inline operator T *() {
//         extern __shared__ int __smem[];
//         return (T *)__smem;
//     }

//     __device__ inline operator const T *() const {
//         extern __shared__ int __smem[];
//         return (T *)__smem;
//     }
// };

// __global__ void map(double *g_idata, double *g_odata, unsigned int n) {
//     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < n) {
//         double value = g_idata[i];
//         g_odata[i] = (value > 0) ? sqrt(sqrt(value)) : 0;  // Compute quadruple root
//     }
// }

// __global__ void reduce(double *g_idata, double *g_odata, unsigned int n) {
//     double *sdata = SharedMemory<double>(); 
//     //extern __shared__ double sdata[];
//     unsigned int tid = threadIdx.x;
//     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

//    /* if (idx < n) {
//         sdata[tid] = g_idata[idx];
//     } else {
//         sdata[tid] = 0;
//     }*/
//      sdata[tid] = (i < n) ? g_idata[i] : 0;

//     __syncthreads();

//     for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
//         if (tid < s) {
//             sdata[tid] += sdata[tid + s];
//         }
//         __syncthreads();
//     }

//     if (tid == 0) {
//         g_odata[blockIdx.x] = sdata[0];
//     }
// }

// #define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
// void gpuAssert(cudaError_t code, const char *file, int line, bool abort = true) {
//     if (code != cudaSuccess) {
//         fprintf(stderr, "GPUassert: %s %s %d\n", cudaGetErrorString(code), file, line);
//         if (abort) exit(code);
//     }
// }
// //int main(int argc, char **argv) {
// extern "C" double sumqroot(int N, int M, int S) {
//     /*if (argc < 4) {
//         fprintf(stderr, "Usage: %s <power_of_two> <mean> <seed>\n", argv[0]);
//         return 1;
//     }

//     int n = 1 << atoi(argv[1]); // assuming at least 8
//    // int n = atoi(argv[1]);
//     int mean = atoi(argv[2]);
//     int seed = atoi(argv[3]);*/

//     int n = 1 << N; // assuming at least 8
//     int mean = M;
//     int seed = S;

//     //int size = 1 << n;
//    // int numThreads = size;
//     int numThreads = n;
//     int maxThreads = 256;  // number of threads per block
//     int numBlocks = n / maxThreads;
//     unsigned int  bytes = n * sizeof(double);
//     int smemSize = maxThreads * sizeof(double);

//     double *h_idata = (double *)malloc(bytes);
//     double *h_odata = (double *)malloc(numBlocks * sizeof(double));
//     double *d_idata, *d_odata, *d_intermediate;

//     checkCudaErrors(cudaMalloc((void **)&d_idata, bytes));
//     checkCudaErrors(cudaMalloc((void **)&d_odata, numBlocks * sizeof(double)));
//     checkCudaErrors(cudaMalloc((void **)&d_intermediate, bytes)); // Ensure it's large enough

//     srand48(seed);
//     for (int i = 0; i < n; i++) {
//         h_idata[i] = -mean * log(drand48());
//     }

//     checkCudaErrors(cudaMemcpy(d_idata, h_idata, bytes, cudaMemcpyHostToDevice));

//     map<<<numBlocks, maxThreads>>>(d_idata, d_intermediate, numThreads);
//     reduce<<<numBlocks, maxThreads, smemSize>>>(d_intermediate, d_odata, numThreads);

//     int s = numBlocks;
//     while (s > 1) {
//        // int nextNumBlocks = (s + maxThreads - 1) / maxThreads;
        
//         reduce<<<(s + maxThreads - 1) / maxThreads, maxThreads, smemSize>>>(d_odata, d_intermediate, s);
//         checkCudaErrors(cudaMemcpy(d_odata, d_intermediate, (s + maxThreads - 1) / maxThreads * sizeof(double), cudaMemcpyDeviceToDevice)); //this line error checks and is IMPORTANT
//         s = (s + maxThreads - 1) / maxThreads;
//         //s = nextNumBlocks;
//     }

//     checkCudaErrors(cudaMemcpy(h_odata, d_odata, sizeof(double), cudaMemcpyDeviceToHost));

//     printf("GPU sum : %f\n\n", h_odata[0]);

//     checkCudaErrors(cudaFree(d_idata));
//     checkCudaErrors(cudaFree(d_odata));
//     checkCudaErrors(cudaFree(d_intermediate));
//     free(h_idata);
//     free(h_odata);

//     return h_odata[0];
// }

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//#include <math.h>
//#include <cuda_runtime.h>

template<class T>
struct SharedMemory {
    __device__ inline operator T *() {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

__global__ void map(double *g_idata, double *g_odata, unsigned int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        double value = g_idata[i];
        g_odata[i] = (value > 0) ? sqrt(sqrt(value)) : 0;  // Compute quadruple root
    }
}

__global__ void reduce(double *g_idata, double *g_odata, unsigned int n) {
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

   /* if (idx < n) {
        sdata[tid] = g_idata[idx];
    } else {
        sdata[tid] = 0;
    }*/
     sdata[tid] = (i < n) ? g_idata[i] : 0;

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

//int main(int argc, char **argv) {
extern "C" double sumqroot(int N, int M, int S) {
    // if (argc < 4) {
    //     fprintf(stderr, "Usage: %s <power_of_two> <mean> <seed>\n", argv[0]);
    //     return 1;
    // }

//     int n = 1 << atoi(argv[1]); // assuming at least 8
//    // int n = atoi(argv[1]);
//     int mean = atoi(argv[2]);
//     int seed = atoi(argv[3]);

    int n = 1 << N; // assuming at least 8
    int mean = M;
    int seed = S;

    //int size = 1 << n;
   // int numThreads = size;
    int numThreads = n;
    int maxThreads = 256;  // number of threads per block
    int numBlocks = n / maxThreads;
    unsigned int  bytes = n * sizeof(double);
    int smemSize = maxThreads * sizeof(double);

    double *h_idata = (double *)malloc(bytes);
    double *h_odata = (double *)malloc(numBlocks * sizeof(double));
    double *d_idata, *d_odata, *d_intermediate;

    checkCudaErrors(hipMalloc((void **)&d_idata, bytes));
    checkCudaErrors(hipMalloc((void **)&d_odata, numBlocks * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&d_intermediate, bytes)); // Ensure it's large enough

    srand48(seed);
    for (int i = 0; i < n; i++) {
        h_idata[i] = -mean * log(drand48());
    }

    checkCudaErrors(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));

    map<<<numBlocks, maxThreads>>>(d_idata, d_intermediate, numThreads);
    reduce<<<numBlocks, maxThreads, smemSize>>>(d_intermediate, d_odata, numThreads);

    int s = numBlocks;
    while (s > 1) {
       // int nextNumBlocks = (s + maxThreads - 1) / maxThreads;
        reduce<<<(s + maxThreads - 1) / maxThreads, maxThreads, smemSize>>>(d_odata, d_intermediate, s);
        checkCudaErrors(hipMemcpy(d_odata, d_intermediate, (s + maxThreads - 1) / maxThreads * sizeof(double), hipMemcpyDeviceToDevice));
        //s = nextNumBlocks;
        s = (s + maxThreads - 1) / maxThreads;
    }

    checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(double), hipMemcpyDeviceToHost));

    printf("GPU sum : %f\n\n", h_odata[0]);

    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));
    checkCudaErrors(hipFree(d_intermediate));
    //free(h_idata);
    //free(h_odata);

    return h_odata[0];
}

